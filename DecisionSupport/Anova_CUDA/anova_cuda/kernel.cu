#include "hip/hip_runtime.h"
// A CUDA-powered tool to quickly rank all possible factorial configurations attending to input scenarios (Weighted-Sum models)
// A set of factors is inferred from input regression models
// Optimization results are exported into XML-formatted file
// Author: Ilya Tuzov, Universitat Politecnica de Valencia

#include "hip/hip_runtime.h"
#include ""


#include <stdio.h>
#include <vector>
#include <regex>
#include <string>
#include <stdint.h>
#include <iostream>
#include <fstream>
#include "tinyxml2.h"
#include <cmath>
#include <math.h>

using namespace std;
using namespace tinyxml2;

#ifndef XMLCheckResult
#define XMLCheckResult(a_eResult) if (a_eResult != XML_SUCCESS) { printf("Error: %i\n", a_eResult); }
#endif

static void HandleError(hipError_t err,
	const char *file,
	int line) {
	if (err != hipSuccess) {
		printf("%s in %s at line %d\n", hipGetErrorString(err), file, line);
	}
}
#define HANDLE_ERROR( err ) (HandleError( err, __FILE__, __LINE__ ))



//-----------------

#define MAX_POINTS_PER_THREAD 4096
#define NBLOCKS 16
#define NTHREADS 512
#define CU_RESULT_LEN NBLOCKS*NTHREADS
#define MAX_MODELS 10
#define MAX_FACTORS 33
#define MAX_INTERACTIONS MAX_FACTORS*8
#define FACTOR_VECTOR_SIZE MAX_FACTORS*MAX_MODELS
#define INTERACTION_VECTOR_SIZE MAX_INTERACTIONS*MAX_MODELS


enum distribution_t {normal, gamma, inverse_gaussian, poisson, undefined};

struct t_estimator_factor {
	string factor;
	int id;
	float value;
	int response_variable_id;
};

struct t_estimator_interaction {
	string factor_1;
	string factor_2;
	int id_1;
	int id_2;
	float value;
	int response_variable_id;
};

struct t_response_variable{
	int id;
	string name;
	string model_filename;
	float intercept;
	float treshold;
	distribution_t dist;
};


struct t_factor_desc{
	string name;
	int id;
};

//returns index of structure for given keyname [0 : i_desc-size()-1]
//if does not exists - returns -1
int get_index(std::vector<t_factor_desc> i_desc, string keyname){
	for (int i = 0; i < i_desc.size(); i++){
		if (i_desc[i].name == keyname) return(i);
	};
	return(-1);
}

//returns id of structure for given keyname [0 : i_desc-size()-1]
//if does not exists - returns -1
int get_id(std::vector<t_factor_desc> i_desc, string keyname){
	for (int i = 0; i < i_desc.size(); i++){
		if (i_desc[i].name == keyname) return(i_desc[i].id);
	};
	return(-1);
}

bool compare_by_name(const t_factor_desc & a, const t_factor_desc & b){
	return(a.name < b.name);
}

//structures optimized for cuda operations
struct cu_factor_estimator{
	int id;
	float value;
	int response_variable_id;
};

struct cu_interaction_estimator{
	int id_1;
	int id_2;
	float value;
	int response_variable_id;
};

struct cu_response_variable{
	int id;
	distribution_t dist;
	float intercept;
	int factor_first;
	int factor_num;
	int interaction_first;
	int interaction_num;
	//computed/updated after first kernell call, entire structure should be re-copied to device constant memory
	float max;
	float min;
};

//Kernel return type
struct cu_thread_result{
	uint64_t max_index;
	uint64_t min_index;
	float max;
	float min;
};

struct input_data_desctiptor{
	int real_models;
	int real_factor_estimators;
	int real_interaction_estimators;
	uint64_t factor_vector_len;
};

struct cu_job_descriptor{
	uint64_t max_index;			//2 ^ input_data_desctiptor.real_factor_estimators
	uint64_t points_per_thread;	//max_index / CU_RESULT_LEN
};



enum minmax_t { v_min, v_max, v_undefined };

struct t_implementation_goal_item {
	minmax_t goal;
	string var_name;
	int var_id;
	float weight;
};

struct t_implementation_goal {
	string goal_name;
	t_implementation_goal_item items[MAX_MODELS];
	int valid_items_num;
};

struct cu_implementation_goal_item {
	minmax_t goal;
	int var_id;
	float weight;
};


//Global variables and functions

cu_factor_estimator *host_factor_vector;
cu_interaction_estimator  *host_interaction_vector;
cu_response_variable *host_response_variables;
input_data_desctiptor host_data_descriptor;
cu_job_descriptor host_job_descriptor;
cu_implementation_goal_item *host_implementation_goal;

__constant__ cu_factor_estimator			cuda_factor_vector[FACTOR_VECTOR_SIZE];
__constant__ cu_interaction_estimator		cuda_interaction_vector[INTERACTION_VECTOR_SIZE];
__constant__ cu_response_variable			cuda_response_variables[MAX_MODELS];
__constant__ input_data_desctiptor			cuda_data_descriptor[1];
__constant__ cu_job_descriptor				cuda_jobdesc[1];
__constant__ cu_implementation_goal_item	cuda_implementation_goal[MAX_MODELS];

//Dummy function required to cope with hipMemcpyToSymbol bug
__constant__ float dev_test_arr[10];
 float tiny_test_arr[10];
 void dummy_wakeup_const_memory(){
	 for (int i = 0; i < 10; i++)tiny_test_arr[i] = float(i) + 1.5;
	 HANDLE_ERROR(hipMemcpyToSymbol(HIP_SYMBOL(dev_test_arr), tiny_test_arr, 10 * sizeof(float)));
 }
 //end dummy code

 //**** AUXILIARY FUNCTIONS (on-host)
 t_estimator_factor * get_estimator_for_model_ptr(std::vector<t_estimator_factor> v_factors, string ifactor, int ivar_id){
	 for (int i = 0; i < v_factors.size(); i++){
		 if ((v_factors[i].factor == ifactor) && (v_factors[i].response_variable_id == ivar_id))
			 return(&v_factors[i]);
	 }
	 return(nullptr);
 }

 t_estimator_factor * get_estimator_any_ptr(std::vector<t_estimator_factor> v_factors, string ifactor){
	 for (int i = 0; i < v_factors.size(); i++){
		 if (v_factors[i].factor == ifactor )
			 return(&v_factors[i]);
	 }
	 return(nullptr);
 }

//******************* HOST Functions
cu_thread_result compute_min_max_on_host(int var_id){
	cu_thread_result res = { 0, 0, 0, 0 };
	bool min_initialized = false, max_initialized = false;
	unsigned int factor_vect[MAX_FACTORS];
	register uint64_t c = 0;
	register uint64_t j = 0;
	uint64_t pp_step = host_job_descriptor.max_index / uint64_t(100);
	uint64_t pp_cnt = 0;
	int pp_progress = 0;
	//
	cout << "\n";
	for (c = 0; c < host_job_descriptor.max_index; c++){
		if (c > pp_cnt){
			cout << "\rCOMPUTE_MIN_MAX progress: " << pp_progress << "%";
			pp_cnt += pp_step;
			pp_progress += 1;
		}
		for (j = 0; j < host_data_descriptor.factor_vector_len; j++)
			factor_vect[j] = unsigned int((c >> j) & uint64_t(1));
		float current = host_response_variables[var_id].intercept;
		//process single-factor estimators
		for (int i = host_response_variables[var_id].factor_first; i < host_response_variables[var_id].factor_first + host_response_variables[var_id].factor_num; i++){
			if (factor_vect[host_factor_vector[i].id] == 1) current += host_factor_vector[i].value;
		}
		//process interactions estimators
		for (int i = host_response_variables[var_id].interaction_first; i < host_response_variables[var_id].interaction_first + host_response_variables[var_id].interaction_num; i++){
			if ((factor_vect[host_interaction_vector[i].id_1] == 1) && (factor_vect[host_interaction_vector[i].id_2] == 1)) current += host_interaction_vector[i].value;
		}
		if (host_response_variables[var_id].dist == gamma) current = 1 / current;
		else if (host_response_variables[var_id].dist == inverse_gaussian) current = 1 / std::sqrtf(current);
		else if (host_response_variables[var_id].dist == poisson) current = std::expf(current);
		if (min_initialized && max_initialized){
			if (current < res.min){
				res.min = current;
				res.min_index = c;
			}
			if (current > res.max){
				res.max = current;
				res.max_index = c;
			}
		}
		else{
			res.min = current; res.max = current;
			min_initialized = true; max_initialized = true;
		}
	};
	return(res);
}


//CUDA FUNCTIONS
__global__ void cu_min_max(int var_id, uint64_t start_index,  cu_thread_result * res_arr)
{
	unsigned int factor_vect[MAX_FACTORS];
	bool min_initialized = false, max_initialized = false;
	cu_thread_result res = { 0, 0, 0, 0 };
	uint64_t idx = uint64_t(blockIdx.x * blockDim.x + threadIdx.x);
	uint64_t offset = idx*cuda_jobdesc[0].points_per_thread;
	uint64_t loc_offset = start_index + offset;

	uint64_t c = 0;
	uint64_t j = 0;
	for (c = loc_offset; (c < loc_offset + cuda_jobdesc[0].points_per_thread) && (c < cuda_jobdesc[0].max_index); c++){
		for (j = 0; j < cuda_data_descriptor[0].factor_vector_len; j++)
			factor_vect[j] = unsigned int((c >> j) & uint64_t(1));		
		float current = cuda_response_variables[var_id].intercept;
		//process single-factor estimators
		for (int i = cuda_response_variables[var_id].factor_first; i < cuda_response_variables[var_id].factor_first + cuda_response_variables[var_id].factor_num; i++){
			if (factor_vect[cuda_factor_vector[i].id] == 1) current += cuda_factor_vector[i].value;
		}
		//process interactions estimators
		for (int i = cuda_response_variables[var_id].interaction_first; i < cuda_response_variables[var_id].interaction_first + cuda_response_variables[var_id].interaction_num; i++){
			if ((factor_vect[cuda_interaction_vector[i].id_1] == 1) && (factor_vect[cuda_interaction_vector[i].id_2] == 1)) current += cuda_interaction_vector[i].value;
		}
		if (cuda_response_variables[var_id].dist == gamma) current = 1 / current;
		else if (cuda_response_variables[var_id].dist == inverse_gaussian) current = 1/std::sqrtf(current);
		else if (cuda_response_variables[var_id].dist == poisson) current = std::expf(current);
		if (min_initialized && max_initialized){
			if (current < res.min){
				res.min = current;
				res.min_index = c;
			}
			if (current > res.max){
				res.max = current;
				res.max_index = c;
			}
		}
		else{
			res.min = current; res.max = current;
			res.min_index = c; res.max_index = c;
			min_initialized = true; max_initialized = true;
		}
	};
	if (offset < cuda_jobdesc[0].max_index){
		res_arr[idx] = res;
		//printf("\nc=%u, min_val =%f, min id = %u, max_val=%f, max id = %u", c, res.min, res.min_index, res.max, res.max_index);
	}

}



__host__ cu_thread_result compute_min_max_on_cuda(int var_id){
	cu_thread_result res = { 0, 0, 0, 0 };
	int dev_result_len = (CU_RESULT_LEN < host_job_descriptor.max_index) ? CU_RESULT_LEN : host_job_descriptor.max_index;
	cu_thread_result * dev_res_arr;
	HANDLE_ERROR(hipMalloc((void**)&dev_res_arr, dev_result_len * sizeof(cu_thread_result)));
	cu_thread_result * host_res_arr = (cu_thread_result*)malloc(sizeof(cu_thread_result)* dev_result_len);

	int iterations = host_job_descriptor.max_index / (dev_result_len*host_job_descriptor.points_per_thread);
	float progress_step=100.0, progress = 0.0;
	if (iterations > 1){
		progress_step = 100.0 / float(iterations);
	};
	for (uint64_t c = 0; c < host_job_descriptor.max_index; c += dev_result_len*host_job_descriptor.points_per_thread){
		progress += progress_step;
		printf("\rCompute min/max on CUDA, variable [%i], progress: %2.2f%%", var_id, progress);
		cu_min_max << < NBLOCKS, NTHREADS >> > (var_id, c, dev_res_arr);
		hipDeviceSynchronize();
		HANDLE_ERROR(hipMemcpy(host_res_arr, dev_res_arr, dev_result_len * sizeof(cu_thread_result), hipMemcpyDeviceToHost));
		if (c == 0)res = host_res_arr[0];
		for (int i = 0; i < dev_result_len; i++){
			if (host_res_arr[i].min < res.min){
				res.min = host_res_arr[i].min;
				res.min_index = host_res_arr[i].min_index;
			};
			if (host_res_arr[i].max > res.max){
				res.max = host_res_arr[i].max;
				res.max_index = host_res_arr[i].max_index;
			};
		}
		//printf("\n\nmin = %f, max = %f", res.min, res.max);
	}
	hipFree(dev_res_arr);
	return(res);
}

//Implementation goals on Cuda
__global__ void cu_implementation_goal(int valid_items_num, uint64_t start_index, cu_thread_result * res_arr)
{
	unsigned int factor_vect[MAX_FACTORS];
	bool min_initialized = false, max_initialized = false;
	cu_thread_result res = { 0, 0, 0, 0 };
	uint64_t idx = uint64_t(blockIdx.x * blockDim.x + threadIdx.x);
	uint64_t offset = idx*cuda_jobdesc[0].points_per_thread;
	uint64_t loc_offset = start_index + offset;

	uint64_t c = 0;
	uint64_t j = 0;
	for (c = loc_offset; (c < loc_offset + cuda_jobdesc[0].points_per_thread) && (c < cuda_jobdesc[0].max_index); c++){
		for (j = 0; j < cuda_data_descriptor[0].factor_vector_len; j++)
			factor_vect[j] = unsigned int((c >> j) & uint64_t(1));

		float goal_current=0;
		for (int t = 0; t < valid_items_num; t++){
			int var_id = cuda_implementation_goal[t].var_id;
			float var_current = cuda_response_variables[var_id].intercept;
			//process single-factor estimators
			for (int i = cuda_response_variables[var_id].factor_first; i < cuda_response_variables[var_id].factor_first + cuda_response_variables[var_id].factor_num; i++){
				if (factor_vect[cuda_factor_vector[i].id] == 1) var_current += cuda_factor_vector[i].value;
			}
			//process interactions estimators
			for (int i = cuda_response_variables[var_id].interaction_first; i < cuda_response_variables[var_id].interaction_first + cuda_response_variables[var_id].interaction_num; i++){
				if ((factor_vect[cuda_interaction_vector[i].id_1] == 1) && (factor_vect[cuda_interaction_vector[i].id_2] == 1)) var_current += cuda_interaction_vector[i].value;
			}
			if (cuda_response_variables[var_id].dist == gamma) var_current = 1 / var_current;
			else if (cuda_response_variables[var_id].dist == inverse_gaussian) var_current = 1 / std::sqrtf(var_current);
			else if (cuda_response_variables[var_id].dist == poisson) var_current = std::expf(var_current);
			//normalize: min or max
			float normalized_current;
			float minv = cuda_response_variables[var_id].min;
			float maxv = cuda_response_variables[var_id].max;
			if (minv > 0){
				if (cuda_implementation_goal[t].goal == v_min) normalized_current = minv / var_current;
				else if (cuda_implementation_goal[t].goal == v_max) normalized_current = var_current/ maxv;
			}
			else if(maxv > 0){
				if (cuda_implementation_goal[t].goal == v_min) normalized_current = (minv+0.1) / (var_current+0.1);
				else if (cuda_implementation_goal[t].goal == v_max) normalized_current = var_current / maxv;
			}
			else{
				normalized_current = 1;
			}
			//*weight and add to goal sum
			goal_current += (normalized_current*cuda_implementation_goal[t].weight);
		}

		if (min_initialized && max_initialized){
			if (goal_current < res.min){
				res.min = goal_current;
				res.min_index = c;
			}
			if (goal_current > res.max){
				res.max = goal_current;
				res.max_index = c;
			}
		}
		else{
			res.min = goal_current; res.max = goal_current;
			res.min_index = c; res.max_index = c;
			min_initialized = true; max_initialized = true;
		}
	};
	if (offset < cuda_jobdesc[0].max_index){
		res_arr[idx] = res;
		//printf("\nc=%u, min_val =%f, min id = %u, max_val=%f, max id = %u", c, res.min, res.min_index, res.max, res.max_index);
	}
}


__host__ cu_thread_result process_implementation_goal_on_cuda(int valid_items_num){
	cu_thread_result res = { 0, 0, 0, 0 };
	for (int i = 0; i < valid_items_num; i++)printf("\n\t\t\tVar_id = %i\tGoal = %i\tWeight = %f", host_implementation_goal[i].var_id, host_implementation_goal[i].goal, host_implementation_goal[i].weight);
	printf("\n");
	int dev_result_len = (CU_RESULT_LEN < host_job_descriptor.max_index) ? CU_RESULT_LEN : host_job_descriptor.max_index;
	cu_thread_result * dev_res_arr;
	HANDLE_ERROR(hipMalloc((void**)&dev_res_arr, dev_result_len * sizeof(cu_thread_result)));
	cu_thread_result * host_res_arr = (cu_thread_result*)malloc(sizeof(cu_thread_result)* dev_result_len);

	int iterations = host_job_descriptor.max_index / (dev_result_len*host_job_descriptor.points_per_thread);
	float progress_step = 100.0, progress = 0.0;
	if (iterations > 1){
		progress_step = 100.0 / float(iterations);
	};
	for (uint64_t c = 0; c < host_job_descriptor.max_index; c += dev_result_len*host_job_descriptor.points_per_thread){
		progress += progress_step;
		printf("\rProcess implementation goal on CUDA, progress: %2.2f%%",  progress);
		cu_implementation_goal << < NBLOCKS, NTHREADS >> > (valid_items_num, c, dev_res_arr);
		hipDeviceSynchronize();
		HANDLE_ERROR(hipMemcpy(host_res_arr, dev_res_arr, dev_result_len * sizeof(cu_thread_result), hipMemcpyDeviceToHost));
		if (c == 0)res = host_res_arr[0];
		for (int i = 0; i < dev_result_len; i++){
			if (host_res_arr[i].min < res.min){
				res.min = host_res_arr[i].min;
				res.min_index = host_res_arr[i].min_index;
			};
			if (host_res_arr[i].max > res.max){
				res.max = host_res_arr[i].max;
				res.max_index = host_res_arr[i].max_index;
			};
		}
		//printf("\n\nmin = %f, max = %f", res.min, res.max);
	}
	hipFree(dev_res_arr);
	return(res);
}

float get_varvalue(int var_id, uint64_t confnum){
	unsigned int factor_vect[MAX_FACTORS];
	register uint64_t j = 0;
	for (j = 0; j < host_data_descriptor.factor_vector_len; j++)
		factor_vect[j] = unsigned int((confnum >> j) & uint64_t(1));
	float res = host_response_variables[var_id].intercept;
	//process single-factor estimators
	for (int i = host_response_variables[var_id].factor_first; i < host_response_variables[var_id].factor_first + host_response_variables[var_id].factor_num; i++){
		if (factor_vect[host_factor_vector[i].id] == 1) res += host_factor_vector[i].value;
	}
	//process interactions estimators
	for (int i = host_response_variables[var_id].interaction_first; i < host_response_variables[var_id].interaction_first + host_response_variables[var_id].interaction_num; i++){
		if ((factor_vect[host_interaction_vector[i].id_1] == 1) && (factor_vect[host_interaction_vector[i].id_2] == 1)) res += host_interaction_vector[i].value;
	}
	if (host_response_variables[var_id].dist == gamma) res = 1 / res;
	else if (host_response_variables[var_id].dist == inverse_gaussian) res = 1 / std::sqrtf(res);
	else if (host_response_variables[var_id].dist == poisson) res = std::expf(res);
	return(res);
}

string get_configvector_csv(std::vector<t_factor_desc> factors, uint64_t config_num){
	string res = "";
	unsigned int factor_vect[MAX_FACTORS];
	int size = factors.size();
	register uint64_t j = 0;
	for (int i = 0; i < size; i++)res += factors[i].name + ",";
	res += "\n";
	for (j = 0; j < host_data_descriptor.factor_vector_len; j++){
		factor_vect[j] = unsigned int((config_num >> j) & uint64_t(1));
		res += std::to_string(factor_vect[j]) + ",\t";
	}
	return(res);
}

string get_fullconfigvector_csv(std::vector<string> full_factor_set, std::vector<t_factor_desc> factors, uint64_t config_num, bool labels){
	unsigned int factor_vect[MAX_FACTORS];
	string part_1="", part_2="";
	register uint64_t j = 0;
	for (j = 0; j < host_data_descriptor.factor_vector_len; j++)
		factor_vect[j] = unsigned int((config_num >> j) & uint64_t(1));
	int bit_cnt = 0;
	for (int i = 0; i < full_factor_set.size(); i++){
		part_1 += full_factor_set[i];
		if (get_id(factors, full_factor_set[i]) < 0){
			part_2 += "-";
		}
		else{
			part_2 += std::to_string(factor_vect[bit_cnt]);
			bit_cnt += 1;
		};
		if (i < full_factor_set.size() - 1){
			part_1 += ",";
			part_2 += ",  ";
		}
	}
	if (labels == true) return(part_1 + "\n" + part_2);
	else return(part_2);
}

string get_fullconfigvector_significant_only_csv(std::vector<string> full_factor_set, std::vector<t_factor_desc> factors, std::vector<t_factor_desc> significant_factors, uint64_t config_num, bool labels){
	unsigned int factor_vect[MAX_FACTORS];
	string part_1 = "", part_2 = "";
	register uint64_t j = 0;
	for (j = 0; j < host_data_descriptor.factor_vector_len; j++)
		factor_vect[j] = unsigned int((config_num >> j) & uint64_t(1));
	int bit_cnt = 0;
	for (int i = 0; i < full_factor_set.size(); i++){
		part_1 += full_factor_set[i];
		if (get_id(factors, full_factor_set[i]) < 0){
			part_2 += "-";
		}
		else{
			if (get_id(significant_factors, full_factor_set[i]) < 0){
				part_2 += "-";
			}
			else{
				part_2 += std::to_string(factor_vect[bit_cnt]);
			}
			bit_cnt += 1;
		};
		if (i < full_factor_set.size() - 1){
			part_1 += ",";
			part_2 += ",  ";
		}
	}
	if (labels == true) return(part_1 + "\n" + part_2);
	else return(part_2);
}

string get_fullconfigvector_varid_specific_csv(std::vector<string> full_factor_set, std::vector<t_estimator_factor> v_factors, int response_variable_id, uint64_t config_num, bool labels){
	unsigned int factor_vect[MAX_FACTORS];
	string part_1 = "", part_2 = "";
	register uint64_t j = 0;
	for (j = 0; j < host_data_descriptor.factor_vector_len; j++)
		factor_vect[j] = unsigned int((config_num >> j) & uint64_t(1));
	int bit_cnt = 0;
	for (int i = 0; i < full_factor_set.size(); i++){
		part_1 += full_factor_set[i];
		if (get_estimator_for_model_ptr(v_factors, full_factor_set[i], response_variable_id) == nullptr){
			part_2 += "-";
		}
		else{
			part_2 += std::to_string(factor_vect[bit_cnt]);
		};
		if (get_estimator_any_ptr(v_factors, full_factor_set[i]) != nullptr) bit_cnt += 1;
		if (i < full_factor_set.size() - 1){
			part_1 += ",";
			part_2 += ",  ";
		}
	}
	if(labels == true) return(part_1 + "\n" + part_2);
	else return(part_2);
}

int main(int argc, char* argv[])
{
	//char *config_file = "C:\\Models_Interactions\\B_14_Join_csv\\itc_config.xml";
	//char *implementation_goals_file = "C:\\Models_Interactions\\implementation_goals.xml";

	char *config_file = argv[1];
	char *implementation_goals_file = argv[2];
	printf("\nINPUT CONFIG FILE: %s", config_file);
	std::vector<t_estimator_factor> v_factors, v_minimized_factors;
	std::vector<t_estimator_interaction> v_interactions, v_minimized_interactions;
	std::vector<t_response_variable> v_response_variables;
	std::vector<t_factor_desc> factor_descriptors;
	std::vector<string> full_factor_set;
	XMLDocument xmlDoc;
	std::vector<t_implementation_goal> v_implementation_goals;


	printf("\nFACTOR_VECTOR_SIZE: %i, sizeof cu_factor_estimator:%i\nINTERACTION_VECTOR_SIZE: %i, sizeof cu_interaction_estimator:%i", FACTOR_VECTOR_SIZE, sizeof(cu_factor_estimator), INTERACTION_VECTOR_SIZE, sizeof(cu_interaction_estimator));

	regex factor_expr("^\\s*([A-Za-z0-9_]+)[\\s,]+([-+]?[0-9]*\\.?[0-9]+([eE][-+]?[0-9]+)?)");
	regex factor_name_expr("([A-Za-z0-9]+)");
	regex interaction_expr("^\\s*([A-Za-z0-9_]+)\\:([A-Za-z0-9_]+)[\\s,]+([-+]?[0-9]*\\.?[0-9]+([eE][-+]?[0-9]+)?)");
	regex decimal_expr("([0-9]+)");
	regex intercept_expr("Intercept.*?[\\s,]+([-+]?[0-9]*\\.?[0-9]+([eE][-+]?[0-9]+)?)");

	//Parse Configuration.xml
	XMLError eResult = xmlDoc.LoadFile(config_file);
	XMLCheckResult(eResult);
	XMLNode * pRoot = xmlDoc.FirstChild();
	if (pRoot == nullptr) printf("\nXML read error: %i", XML_ERROR_FILE_READ_ERROR);
	XMLElement * Xitem = pRoot->FirstChildElement("models")->FirstChildElement("responce_variable");
	string model_path = pRoot->FirstChildElement("models")->Attribute("path");
	int id = 0;
	while (Xitem != nullptr){
		//printf("\n%s", Xitem->Attribute("file"));
		struct t_response_variable c;
		c.id = id;
		c.name = Xitem->Attribute("name");
		c.model_filename = model_path + Xitem->Attribute("file");
		c.treshold = stof(Xitem->Attribute("value_treshold"));
		string dist = Xitem->Attribute("distribution");
		if (strcmp(dist.c_str(), "normal") == 0) c.dist = normal;
		else if (strcmp(dist.c_str(), "gamma") == 0) c.dist = gamma;
		else if (strcmp(dist.c_str(), "inverse gaussian") == 0) c.dist = inverse_gaussian;
		else if (strcmp(dist.c_str(), "poisson") == 0) c.dist = poisson;
		else c.dist = undefined;
		v_response_variables.push_back(c);
		Xitem = Xitem->NextSiblingElement("responce_variable");
		id++;
	}
	string fullfactorstring = pRoot->FirstChildElement("FactorialDesign")->Attribute("FullFactorSet");
	if (strcmp(fullfactorstring.c_str(), "") != 0){
		smatch match;
		while (regex_search(fullfactorstring, match, factor_name_expr)){
			full_factor_set.push_back(match[1]);
			fullfactorstring = match.suffix();
		}
	}

	//Parse Implementation_goals.xml
	eResult = xmlDoc.LoadFile(implementation_goals_file);
	XMLCheckResult(eResult);
	pRoot = xmlDoc.FirstChild();
	if (pRoot == nullptr) printf("\nXML read error: %i", XML_ERROR_FILE_READ_ERROR);
	Xitem = pRoot->FirstChildElement("goal");
	while (Xitem != nullptr){
		struct t_implementation_goal c;
		c.goal_name = Xitem->Attribute("name");
		XMLElement * V_item = Xitem->FirstChildElement("variable");
		c.valid_items_num = 0;
		while (V_item != nullptr){
			struct t_implementation_goal_item v;
			v.var_name = V_item->Attribute("name");
			v.weight = stof(V_item->Attribute("weight"));
			string buf = V_item->Attribute("goal");
			if (strcmp(buf.c_str(), "max") == 0) v.goal = v_max;
			else if (strcmp(buf.c_str(), "min") == 0) v.goal = v_min;
			else v.goal = v_undefined;
			c.items[c.valid_items_num] = v;
			c.valid_items_num += 1;
			V_item = V_item->NextSiblingElement("variable");
		}
		v_implementation_goals.push_back(c);
		Xitem = Xitem->NextSiblingElement("goal");
	}
	

	//1. Read regression model from file into v_factors and v_interactions
	std::string line;
	for (int i = 0; i<v_response_variables.size(); i++){
		std::ifstream rm_file(v_response_variables[i].model_filename);
		if (rm_file.is_open()){
			while (std::getline(rm_file, line)){
				try{
					smatch match;
					smatch idx_match;
					if (regex_search(line, match, intercept_expr) && match.size() > 1){
						v_response_variables[i].intercept = stof(match[1].str());
						printf("\nIntercept[%i]: %f", v_response_variables[i].id, v_response_variables[i].intercept);
					}
					else if (regex_search(line, match, factor_expr) && match.size() > 2){
						smatch local_match;
						struct t_estimator_factor c;
						c.factor = match[1].str();
						regex_search(c.factor, local_match, factor_name_expr);
						c.factor = local_match[1];
						c.value = stof(match[2].str());
						c.response_variable_id = v_response_variables[i].id;
						v_factors.push_back(c);
					}
					else if (regex_search(line, match, interaction_expr) && match.size() > 3){
						smatch local_match;
						struct t_estimator_interaction c;
						c.factor_1 = match[1].str();
						c.factor_2 = match[2].str();

						regex_search(c.factor_1, local_match, factor_name_expr);
						c.factor_1 = local_match[1];
						regex_search(c.factor_2, local_match, factor_name_expr);
						c.factor_2 = local_match[1];

						c.value = stof(match[3].str());
						c.response_variable_id = v_response_variables[i].id;
						v_interactions.push_back(c);
					}
				}
				catch (regex_error & e){
				}
			}
			rm_file.close();
		}
		else cout << "Unable to open file";
	}

	for (int i = 0; i < v_factors.size(); i++) printf("\nVarId: %i, Factor: %s, value = %g", v_factors[i].response_variable_id, v_factors[i].factor.c_str(), v_factors[i].value);
	for (int i = 0; i < v_interactions.size(); i++) printf("\nVarId: %i, Interaction: %s : %s, value = %g", v_interactions[i].response_variable_id, v_interactions[i].factor_1.c_str(), v_interactions[i].factor_2.c_str(), v_interactions[i].value);
	//Minimize model (remove factors/interaction with impact smaller than treshold (to speed-up computations)
	for (int i = 0; i < v_factors.size(); i++){
		if (fabs(v_factors[i].value) >= v_response_variables[v_factors[i].response_variable_id].treshold) v_minimized_factors.push_back(v_factors[i]);
	}
	for (int i = 0; i < v_interactions.size(); i++){
		if (fabs(v_interactions[i].value) >= v_response_variables[v_interactions[i].response_variable_id].treshold) v_minimized_interactions.push_back(v_interactions[i]);
	}
	printf("\n\n\t\tMINIMIZED MODEL:\n\n");
	for (int i = 0; i < v_minimized_factors.size(); i++) printf("\nVarId: %i, Factor: %s, value = %g", v_minimized_factors[i].response_variable_id, v_minimized_factors[i].factor.c_str(), v_minimized_factors[i].value);
	for (int i = 0; i < v_minimized_interactions.size(); i++) printf("\nVarId: %i, Interaction: %s : %s, value = %g", v_minimized_interactions[i].response_variable_id, v_minimized_interactions[i].factor_1.c_str(), v_minimized_interactions[i].factor_2.c_str(), v_minimized_interactions[i].value);

	for (int i = 0; i < v_minimized_factors.size(); i++){
		if (get_index(factor_descriptors, v_minimized_factors[i].factor) < 0){
			struct t_factor_desc c = { v_minimized_factors[i].factor, 0 };
			factor_descriptors.push_back(c);
		}
	};
	for (int i = 0; i < v_minimized_interactions.size(); i++){
		if (get_index(factor_descriptors, v_minimized_interactions[i].factor_1) < 0){
			struct t_factor_desc c = { v_minimized_interactions[i].factor_1, 0 };
			factor_descriptors.push_back(c);
		}
		if (get_index(factor_descriptors, v_minimized_interactions[i].factor_2) < 0){
			struct t_factor_desc c = { v_minimized_interactions[i].factor_2, 0 };
			factor_descriptors.push_back(c);
		}
	};
	sort(factor_descriptors.begin(), factor_descriptors.end(), compare_by_name);
	for (int i = 0; i < factor_descriptors.size(); i++)factor_descriptors[i].id = i;

	for (int i = 0; i < v_minimized_factors.size(); i++){
		int idx = get_id(factor_descriptors, v_minimized_factors[i].factor);
		if (idx >= 0) v_minimized_factors[i].id = idx;
		else  printf("\nERROR: Index not found for factor: %s", v_minimized_factors[i].factor);
	};
	for (int i = 0; i < v_minimized_interactions.size(); i++){
		int idx_1 = get_id(factor_descriptors, v_minimized_interactions[i].factor_1);
		int idx_2 = get_id(factor_descriptors, v_minimized_interactions[i].factor_2);
		if (idx_1 >= 0 && idx_2 >= 0){ v_minimized_interactions[i].id_1 = idx_1; v_minimized_interactions[i].id_2 = idx_2; }
		else  printf("\nERROR: Index not found for interaction: %s : %s", v_minimized_interactions[i].factor_1, v_minimized_interactions[i].factor_2);
	};
	for (int i = 0; i < factor_descriptors.size(); i++)printf("\n\tFactor ID: %i, Name: %s", factor_descriptors[i].id, factor_descriptors[i].name.c_str());


	//********** MEMORY ALLOCATION ******************//
	host_factor_vector = (cu_factor_estimator*)malloc(sizeof(cu_factor_estimator)* FACTOR_VECTOR_SIZE);
	host_interaction_vector = (cu_interaction_estimator*)malloc(sizeof(cu_interaction_estimator)* INTERACTION_VECTOR_SIZE);
	host_response_variables = (cu_response_variable*)malloc(sizeof(cu_response_variable)* MAX_MODELS);
	host_data_descriptor = { v_response_variables.size(), v_minimized_factors.size(), v_minimized_interactions.size(), factor_descriptors.size() };
	host_job_descriptor = { uint64_t(1) << (factor_descriptors.size()), (uint64_t(1) << (factor_descriptors.size())) / uint64_t(CU_RESULT_LEN) };
	if (host_job_descriptor.points_per_thread < 1){
		host_job_descriptor.points_per_thread = 1;
	}
	else if (host_job_descriptor.points_per_thread > MAX_POINTS_PER_THREAD){
		host_job_descriptor.points_per_thread = MAX_POINTS_PER_THREAD;
	}


	for (int i = 0; i < host_data_descriptor.real_factor_estimators; i++){
		host_factor_vector[i].id = v_minimized_factors[i].id;
		host_factor_vector[i].response_variable_id = v_minimized_factors[i].response_variable_id;
		host_factor_vector[i].value = v_minimized_factors[i].value;
	};
	for (int i = 0; i < host_data_descriptor.real_interaction_estimators; i++){
		host_interaction_vector[i].id_1 = v_minimized_interactions[i].id_1;
		host_interaction_vector[i].id_2 = v_minimized_interactions[i].id_2;
		host_interaction_vector[i].response_variable_id = v_minimized_interactions[i].response_variable_id;
		host_interaction_vector[i].value = v_minimized_interactions[i].value;
	};
	for (int i = 0; i < host_data_descriptor.real_models; i++){
		host_response_variables[i].factor_first = 0;
		host_response_variables[i].interaction_first = 0;
		host_response_variables[i].id = v_response_variables[i].id;
		host_response_variables[i].intercept = v_response_variables[i].intercept;
		host_response_variables[i].dist = v_response_variables[i].dist;
		//factors
		for (int j = 0; j < host_data_descriptor.real_factor_estimators; j++){
			if (host_factor_vector[j].response_variable_id == host_response_variables[i].id){
				host_response_variables[i].factor_first = j;
				break;
			}
		};
		int len = 0;
		for (int j = host_response_variables[i].factor_first; j < host_data_descriptor.real_factor_estimators; j++){
			if (host_factor_vector[j].response_variable_id != host_response_variables[i].id){
				len = j - host_response_variables[i].factor_first;
				break;
			}
		};
		if (len == 0){
			if (host_response_variables[i].factor_first == 0 && host_response_variables[i].id != 0)len = 0;
			else len = host_data_descriptor.real_factor_estimators - host_response_variables[i].factor_first;
		}
		host_response_variables[i].factor_num = len;
		//interactions
		for (int j = 0; j < host_data_descriptor.real_interaction_estimators; j++){
			if (host_interaction_vector[j].response_variable_id == host_response_variables[i].id){
				host_response_variables[i].interaction_first = j;
				break;
			}
		};
		len = 0;
		for (int j = host_response_variables[i].interaction_first; j < host_data_descriptor.real_interaction_estimators; j++){
			if (host_interaction_vector[j].response_variable_id != host_response_variables[i].id){
				len = j - host_response_variables[i].interaction_first;
				break;
			}
		};
		if (len == 0){ 
			if (host_response_variables[i].interaction_first == 0 && host_response_variables[i].id!=0) len = 0;
			else len = host_data_descriptor.real_interaction_estimators - host_response_variables[i].interaction_first;
		}
		host_response_variables[i].interaction_num = len;
	};

	for (int i = 0; i < host_data_descriptor.real_models; i++){
		printf("\n\nmodel [%i], factor_first [%i], factor_len [%i], interaction_first [%i], interaction_len [%i], intercept [%g]", host_response_variables[i].id, host_response_variables[i].factor_first, host_response_variables[i].factor_num, host_response_variables[i].interaction_first, host_response_variables[i].interaction_num, host_response_variables[i].intercept);
	}

	//link variables in implementation goals to the models by variable id
	for (int i = 0; i < v_implementation_goals.size(); i++){
		for (int j = 0; j < v_implementation_goals[i].valid_items_num; j++){
			for (int c = 0; c < v_response_variables.size(); c++){
				if (v_implementation_goals[i].items[j].var_name == v_response_variables[c].name){
					v_implementation_goals[i].items[j].var_id = v_response_variables[c].id;
				}
			}
		}
	}


	dummy_wakeup_const_memory();
	HANDLE_ERROR(hipMemcpyToSymbol(HIP_SYMBOL(cuda_factor_vector), host_factor_vector, sizeof(cu_factor_estimator)* FACTOR_VECTOR_SIZE));
	HANDLE_ERROR(hipMemcpyToSymbol(HIP_SYMBOL(cuda_interaction_vector), host_interaction_vector, sizeof(cu_interaction_estimator)* INTERACTION_VECTOR_SIZE));
	HANDLE_ERROR(hipMemcpyToSymbol(HIP_SYMBOL(cuda_response_variables), host_response_variables, sizeof(cu_response_variable)* MAX_MODELS,0,hipMemcpyHostToDevice));
	HANDLE_ERROR(hipMemcpyToSymbol(HIP_SYMBOL(cuda_data_descriptor), &host_data_descriptor, sizeof(input_data_desctiptor)));
	HANDLE_ERROR(hipMemcpyToSymbol(HIP_SYMBOL(cuda_jobdesc), &host_job_descriptor, sizeof(cu_job_descriptor)));

	//MCDM STARTS HERE
	string summary_file_name = model_path + "MCDM_summary.xml";
	FILE* fp = std::fopen(summary_file_name.c_str(), "w");
	fprintf(fp, "<Summary \n\tJoined_Factor_Set=\"");
	for (int i = 0; i < factor_descriptors.size(); i++)fprintf(fp, "%s,", factor_descriptors[i].name.c_str()); 
	fprintf(fp, "\"\n\tFull_Factor_Set=\"");
	for (int i = 0; i < full_factor_set.size(); i++)fprintf(fp, "%s,", full_factor_set[i].c_str());
	fprintf(fp, "\"\n\t>");

	for (int i = 0; i < host_data_descriptor.real_models; i++){
		printf("\n\n");
		int m = host_response_variables[i].id;
		//cu_thread_result freq = compute_min_max_on_host(i);
		cu_thread_result minmax = compute_min_max_on_cuda(m);
		host_response_variables[m].max = (minmax.max > v_response_variables[m].treshold) ? minmax.max : 0;
		host_response_variables[m].min = (minmax.min > v_response_variables[m].treshold) ? minmax.min : 0;
		//string config_min = get_configvector_csv(factor_descriptors, minmax.min_index);
		//string config_max = get_configvector_csv(factor_descriptors, minmax.max_index);
		//string config_min = get_fullconfigvector_csv(full_factor_set, factor_descriptors, minmax.min_index);
		//string config_max = get_fullconfigvector_csv(full_factor_set, factor_descriptors, minmax.max_index);
		string config_min = get_fullconfigvector_varid_specific_csv(full_factor_set, v_minimized_factors, m, minmax.min_index, false);
		string config_max = get_fullconfigvector_varid_specific_csv(full_factor_set, v_minimized_factors, m, minmax.max_index, false);

		fprintf(fp, "\n\n\t<Model\n\t\tName = \"%s\"\n\t\tMin_Val= \"%.2f\"\n\t\tMax_Val= \"%.2f\"", v_response_variables[m].name.c_str(), host_response_variables[m].min, host_response_variables[m].max);
		fprintf(fp, "\n\t\tConfig_Min=\"%s\"\n\t\tConfig_Max=\"%s\"\n\t/>", config_min.c_str(), config_max.c_str());
		cout << "\nModel[" << m << "] = " << v_response_variables[m].name << "\n\tmin_val= " << host_response_variables[m].min << " (conf: " << minmax.min_index << "),\n\tmax_val= " << host_response_variables[m].max << " (conf: " << minmax.max_index << ")";
		//cout << "\nConfig Min:\n" << config_min << "\nConfig Max:\n" << config_max;
	};
	HANDLE_ERROR(hipMemcpyToSymbol(HIP_SYMBOL(cuda_response_variables), host_response_variables, sizeof(cu_response_variable)* MAX_MODELS, 0, hipMemcpyHostToDevice));
	printf("\n\n");




	//Process implementation goals
	host_implementation_goal = (cu_implementation_goal_item*)malloc(sizeof(cu_implementation_goal_item)*MAX_MODELS);
	for (int i = 0; i<v_implementation_goals.size(); i++){
		int valid_items_num = v_implementation_goals[i].valid_items_num;
		for (int j = 0; j<valid_items_num; j++){
			host_implementation_goal[j].goal = v_implementation_goals[i].items[j].goal;
			host_implementation_goal[j].var_id = v_implementation_goals[i].items[j].var_id;
			host_implementation_goal[j].weight = v_implementation_goals[i].items[j].weight;
		};
		HANDLE_ERROR(hipMemcpyToSymbol(HIP_SYMBOL(cuda_implementation_goal), host_implementation_goal, sizeof(cu_implementation_goal_item)*MAX_MODELS));
		printf("\n\nProcess Implementation goal: %s", v_implementation_goals[i].goal_name.c_str());
		cu_thread_result minmax = process_implementation_goal_on_cuda(valid_items_num);
		
		std::vector<t_factor_desc> significant_factors;
		for (int j = 0; j < valid_items_num; j++){
			if (v_implementation_goals[i].items[j].weight > 0){
				int var_id = v_implementation_goals[i].items[j].var_id;
				//single factors
				for (int x = 0; x < v_minimized_factors.size(); x++){
					if (v_minimized_factors[x].response_variable_id == var_id){
						int tx = get_index(factor_descriptors, v_minimized_factors[x].factor);
						if (get_index(significant_factors, v_minimized_factors[x].factor) < 0){
							significant_factors.push_back(factor_descriptors[tx]);
						}
					}
				}
				//interactions
				for (int x = 0; x < v_minimized_interactions.size(); x++){
					if (v_minimized_interactions[x].response_variable_id == var_id){
						int tx1 = get_index(factor_descriptors, v_minimized_interactions[x].factor_1);
						int tx2 = get_index(factor_descriptors, v_minimized_interactions[x].factor_2);
						if (get_index(significant_factors, v_minimized_interactions[x].factor_1) < 0){
							significant_factors.push_back(factor_descriptors[tx1]);
						}
						if (get_index(significant_factors, v_minimized_interactions[x].factor_2) < 0){
							significant_factors.push_back(factor_descriptors[tx2]);
						}
					}
				}
			}
		}
		sort(significant_factors.begin(), significant_factors.end(), compare_by_name);


		string config_min = get_fullconfigvector_significant_only_csv(full_factor_set, factor_descriptors, significant_factors, minmax.min_index, false);
		string config_max = get_fullconfigvector_significant_only_csv(full_factor_set, factor_descriptors, significant_factors, minmax.max_index, false);
		cout << "\n\t\tMin_score: " << minmax.min << " (conf: " << minmax.min_index << ")\n\t\tMax_score: " << minmax.max << " (conf: " << minmax.max_index << ")";

		//fprintf(fp, "\n\n\t<Goal \n\t\tname = \"%s\"\n\t\tBest_Score = \"%.5f\"\n\t\tWorst_Score = \"%.5f\"\n\t\tBest_Config =  \"%s\"\n\t\tWorst_Config = \"%s\"\n\t>", v_implementation_goals[i].goal_name.c_str(), minmax.max, minmax.min, config_max.c_str(), config_min.c_str());
		fprintf(fp, "\n\n\t<Goal name = \"%s\">", v_implementation_goals[i].goal_name.c_str());
		fprintf(fp, "\n\t\t<Best\n\t\t\tScore = \"%.2f\"\n\t\t\tConfig = \"%s\"\n\t\t>", minmax.max, config_max.c_str());
		for (int j = 0; j < valid_items_num; j++){
			float val = get_varvalue(v_implementation_goals[i].items[j].var_id, minmax.max_index);
			fprintf(fp, "\n\t\t\t<Variable name = \"%s\" value = \"%.2f\"/>", v_implementation_goals[i].items[j].var_name.c_str(), val);
		}
		fprintf(fp, "\n\t\t</Best>\n\t\t<Worst\n\t\t\tScore = \"%.2f\"\n\t\t\tConfig = \"%s\"\n\t\t>", minmax.min, config_min.c_str());
		for (int j = 0; j < valid_items_num; j++){
			float val = get_varvalue(v_implementation_goals[i].items[j].var_id, minmax.min_index);
			fprintf(fp, "\n\t\t\t<Variable name = \"%s\" value = \"%.2f\"/>", v_implementation_goals[i].items[j].var_name.c_str(), val);
		}

		fprintf(fp,"\n\t\t</Worst>\n\t</Goal>");
	}

	fprintf(fp, "\n\n</Summary>\n");
	fclose(fp);
	return 0;
}

